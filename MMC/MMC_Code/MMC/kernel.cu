#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""


#include "NewtonRaphson.h"
#include "LU.h"
#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{

    std::cout << "Finding root of test function\n";
    NRSolver nr;
    Func myfunc;
    std::cout << "x = " << nr.root(myfunc, 0.0, 1.0) << '\n';

    // Test LU decomposition with small matrix and known result
    int N = 4;
    double** inmat{};
    inmat = new double* [N];
    std::cout << "Test matrix:\n";
    for (int i = 0; i < N; i++)
    {
        inmat[i] = new double[N];
        for (int j = 0; j < N; j++)
        {
            inmat[i][j] = (j == (N - i) - 1) ? -1.0 : 1.0;
            std::cout << inmat[i][j] << " ";
        }
        std::cout << "\n";
    }
    LU alu(inmat, N);
    // Get the inverse of the matrix (overwrites the input matrix)
    inmat = alu.inverse();
    // Print the result
    std::cout << "Inverse of matrix:\n";
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            std::cout << inmat[i][j] << " ";
        }
        std::cout << "\n";
    }

    // Now, do it on the GPU
    
    double* inlist = new double[N * N];
    int ii = N - 1;
    for (int i = 0; i < N * N; i++)
    {
        if ((i % ii) == 0)
        {
            inlist[i] = -1.0;
            ii--;
            std::cout << inlist[i] << " ";
        }
        else
        {
            inlist[i] = 1.0;
            std::cout << inlist[i] << " ";
        }
    }
    std::cout << "\n";
    
    return 0;

    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
